#include "defs.h"
#include "Matcher.h"
#include "sifts.h"
#include "Gridder.h"

int main(int argc,char *argv[])
{

  if(argc == 5) //Do match-graph construction for given list
	{

		std::vector<std::string> list_img;
		std::ifstream img_files;
		std::string line;
		img_files.open(argv[1]);

		while(!img_files.eof())
		{
			img_files >> line;
			list_img.push_back(line);
		}

		img_files.close();

		std::vector<std::string> list_key;
		std::ifstream key_files;
		key_files.open(argv[2]);

		while(!key_files.eof())
		{
			key_files >> line;
			list_key.push_back(line);
		}

		key_files.close();



		std::vector<sift_img> allSifts(list_key.size());
		std::vector<std::vector<unsigned char> > allDescriptors(list_key.size(),std::vector<unsigned char> (1));
		std::vector<std::vector<SiftGPU::SiftKeypoint> > allKeys(list_key.size(),std::vector<SiftGPU::SiftKeypoint> (1));


		SiftGPU::SiftKeypoint **h_allKeys = (SiftGPU::SiftKeypoint **)malloc(sizeof(SiftGPU::SiftKeypoint*)*list_key.size());
		unsigned char **h_allDesc = (unsigned char **)malloc(sizeof(unsigned char *) * list_key.size());

		std::cout << list_key.size() << std::endl;

		for(int i = 0 ; i < list_key.size(); i++)
		{
			std::cout << list_key[i] << "\n" ; 
			allSifts[i].SetSift((char *)list_key[i].c_str(),allKeys[i],allDescriptors[i]);
			allSifts[i].init(allSifts[i].num_keys,(char *)list_img[i].c_str());
			hipMalloc((void **)&h_allKeys[i],allSifts[i].num_keys*sizeof(SiftGPU::SiftKeypoint));
			hipMemcpy(h_allKeys[i], allKeys[i].data(), sizeof(SiftGPU::SiftKeypoint)*allSifts[i].num_keys, hipMemcpyHostToDevice); 
			hipMalloc((void **)&h_allDesc[i],allSifts[i].num_keys*128*sizeof(unsigned char));
			hipMemcpy(h_allDesc[i], allDescriptors[i].data(), sizeof(unsigned char)*allSifts[i].num_keys*128, hipMemcpyHostToDevice); 
			printf("reading id %d\n",i);
		}

		unsigned char **d_allDesc;
		hipMalloc((void **)&d_allDesc,sizeof(unsigned char*) * list_key.size());

		unsigned char **d_allKeys;
		hipMalloc((void **)&d_allKeys,sizeof(SiftGPU::SiftKeypoint*) * list_key.size());

		hipMemcpy(d_allDesc, h_allDesc, sizeof(unsigned char*) * list_key.size(), hipMemcpyHostToDevice);
		hipMemcpy(d_allKeys, h_allKeys, sizeof(SiftGPU::SiftKeypoint*) * list_key.size(), hipMemcpyHostToDevice);

		std::ifstream bundler;
		bundler.open (argv[3]);

		float f_matrix[9];
		int id1,id2;
		int i = 1;
		double ttime = 0;
		
		std::ofstream myfile;
		myfile.open (argv[4]);

		while(!bundler.eof())
		{
			bundler >> id1 >> id2 >> f_matrix[0] >> f_matrix[1] >> f_matrix[2] >> f_matrix[3] >> f_matrix[4] >> f_matrix[5] >> f_matrix[6] >> f_matrix[7] >> f_matrix[8] ; 
		
			std::cout << i << " " << id1 << " " << id2 << "\n";
			i++;
			Matcher new_matcher;

			ttime += new_matcher.matchImagePair(allSifts[id1],allSifts[id2],f_matrix,h_allKeys[id1],h_allDesc[id1],h_allKeys[id2],h_allDesc[id2]);	
	
			for(int i = 0 ; i < new_matcher.matches.size(); i++)
			{
				int sid1 = new_matcher.matches[i].first ;
				int sid2 = new_matcher.matches[i].second ;
				if(id1 >= list_key.size() || id2 >= list_key.size())
					printf("image id invalid\n");
				else if(sid1 >= allSifts[id1].num_keys)
					printf("%d %d\n",id1,sid1);
				else if (sid2 >= allSifts[id2].num_keys)
					printf("%d %d\n",id2,sid2);
				myfile << id1 << " " << sid1 << " " << allKeys[id1][sid1].x-(allSifts[id1].width/2) << " " << (allSifts[id1].height/2)-allKeys[id1][sid1].y << " " << id2 << " " << sid2 << " " << allKeys[id2][sid2].x -(allSifts[id2].width/2)<< " " << (allSifts[id2].height/2)-allKeys[id2][sid2].y << "\n";
			}
			std::cout << "Total time : " << ttime << std::endl;
			std::cout << "found\n" ;

		}

		myfile.close();
		bundler.close();	
		std::cout << "Total time : " << ttime << std::endl;



	}
	return 0;
}
