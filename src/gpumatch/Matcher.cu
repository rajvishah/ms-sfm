#include "hip/hip_runtime.h"
#include "defs.h"
#include "sifts.h"
#include "Gridder.h"
#include "Matcher.h"

#include <thrust/extrema.h>

//============================== DESCRIPTION =====================================
//   
//   This code deals with matching sifts between two images                       
//   This code implements the algorithm for GPU Matching. It is assumed that      
//   F-matrix is computed from the coarse k% features and remaining features are  
//   matched using this algorithm. We will match SIFT                             
//   features in the query/source image to those in the target image.             
//                                                                                
//================================================================================

//============================ Declare textures ==================================


texture <unsigned char, 1, hipReadModeElementType> tex_sourceDescriptor;
texture <unsigned char, 1, hipReadModeElementType> tex_targetDescriptor;


void Matcher::visualizeMatches(sift_img source,sift_img target,std::vector<std::pair<int,int> > matches,std::vector<SiftGPU::SiftKeypoint> &keys1,std::vector<SiftGPU::SiftKeypoint> &keys2) 
{
	const char* saveAs = NULL;
	cv::Mat queryImage = source.img;
	int qWidth = source.width;
	int qHeight = source.height;

	cv::Mat referenceImage = target.img;
	int rWidth = target.width;
	int rHeight = target.height;

	int canvasWidth = qWidth + rWidth;
	int canvasHeight = qHeight > rHeight ? qHeight : rHeight;
	cv::Mat canvas(canvasHeight, canvasWidth, CV_8UC3, cv::Scalar(0,0,0));

	cv::Rect roi1 = cv::Rect(0,0,qWidth,qHeight);
	cv::Mat canvas_roi1 = canvas(roi1);
	queryImage.copyTo(canvas_roi1);

	cv::Rect roi2 = cv::Rect(qWidth,0,rWidth,rHeight);
	cv::Mat canvas_roi2 = canvas(roi2);
	referenceImage.copyTo(canvas_roi2);

	for(int i=0; i < matches.size(); i++) {
		std::pair<int,int> p = matches[i];
		cv::Point pt1 = cv::Point(keys1[p.first].x, keys1[p.first].y);
		cv::Point pt2 = cv::Point(keys2[p.second].x + qWidth, keys2[p.second].y);

		cv::circle(canvas, pt1, 2, cv::Scalar(0,255,0), 4);
		cv::circle(canvas, pt2, 2, cv::Scalar(0,255,0), 4);
		cv::line(canvas, pt1, pt2, cv::Scalar(0,255,0), 4);
	}

	cv::namedWindow("FeatureMatches",cv::WINDOW_NORMAL);
	imshow("FeatureMatches", canvas);
	cv::waitKey();
	if(saveAs != NULL) {
		imwrite( saveAs, canvas );
	}
}


struct cast 
{
	__host__ __device__
		unsigned char operator()(float x) { return ((unsigned char)(512*x)); }
};

__global__ void computeGridID(SiftGPU::SiftKeypoint *keys, int *gridID1,int *gridID2,int *gridID3,int *gridID4, int num_keys,int gridSize,int numXGrids1,int halfSize,int numXGrids2,int numGrids,int numGridsXOv,int numGridsYOv)
{	
	//=========== Function to compute gridID given siftID ===========

	// Find data location to be processed by current thread
	int siftID = blockIdx.x*blockDim.x + threadIdx.x;	
	if(siftID < num_keys) 
	{
		double simpleX = ((double)keys[siftID].x/(double)gridSize);
		double simpleY = ((double)keys[siftID].y/(double)gridSize);

		double ov = (double)halfSize/gridSize;
		gridID1[siftID] = floor(simpleY)*numXGrids1 + floor(simpleX);
		gridID2[siftID] = floor(simpleY)*numXGrids2 + floor(simpleX - ov);
		gridID3[siftID] = floor(simpleY - ov)*numXGrids1 + floor(simpleX);
		gridID4[siftID] = floor(simpleY -ov)*numXGrids2 + floor(simpleX - ov);
		
		if(gridID2[siftID] < 0)
			gridID2[siftID] = 0;
		if(gridID3[siftID] < 0)
			gridID3[siftID] = 0;
		if(gridID4[siftID] < 0)
			gridID4[siftID] = 0;

		// gridID corresponding to the given siftID is computed
	}
	__syncthreads();

}


__global__ void findTopOfBucket(int *d_gridID,int *d_gridID2,int *d_gridID3,int *d_gridID4, int num_elements,int *d_topOfBucket_ptr,int *d_topOfBucket2_ptr,int *d_topOfBucket3_ptr,int *d_topOfBucket4_ptr,int numGrids,int numGridsXOv,int numGridsYOv,int numGridsXYOv)
{
	// Find data location to be processed by current thread
	int tidx = blockIdx.x*blockDim.x + threadIdx.x;	
	if(tidx < num_elements && tidx != 0) 
	{
		d_topOfBucket_ptr[d_gridID[0]] = 0;
		d_topOfBucket2_ptr[d_gridID2[0]] = 0;
		d_topOfBucket3_ptr[d_gridID3[0]] = 0;
		d_topOfBucket4_ptr[d_gridID4[0]] = 0;
		
		if (d_gridID[tidx] != d_gridID[tidx-1])
			d_topOfBucket_ptr[d_gridID[tidx]] = tidx;
		if (d_gridID2[tidx] != d_gridID2[tidx-1])
			d_topOfBucket2_ptr[d_gridID2[tidx]] = tidx;
		if (d_gridID3[tidx] != d_gridID3[tidx-1])
			d_topOfBucket3_ptr[d_gridID3[tidx]] = tidx;
		if (d_gridID4[tidx] != d_gridID4[tidx-1])
			d_topOfBucket4_ptr[d_gridID4[tidx]] = tidx;

	}
	__syncthreads();
}

__global__ void	findNumSift(int *gridID,int *topOfBucket,int *gridID2,int *topOfBucket2,int *gridID3,int *topOfBucket3,int *gridID4,int *topOfBucket4,int numGrids,int numGridsXOv,int numGridsYOv,int numGridsXYOv,int *numSift,int *numSift2,int *numSift3,int *numSift4)
{
	//===========  Find number of target sifts in each grid ===========

	// Find data location to be processed by current thread
	int gridid = blockIdx.x*blockDim.x + threadIdx.x;	
	int i,numsift = 0;
	if(gridid < numGrids)
	{
		if(topOfBucket[gridid]!=-1)
		{
			for(i = topOfBucket[gridid]; gridID[i] == gridid; i++)
				numsift++;
			numSift[gridid] = numsift;
		}
		else
			numSift[gridid] = 0;
	}
	if(gridid < numGridsXOv)
	{
		if(topOfBucket2[gridid]!=-1)
		{
			numsift = 0;
			for(i = topOfBucket2[gridid]; gridID2[i] == gridid; i++)
				numsift++;
			numSift2[gridid] = numsift;
		}
		else
			numSift2[gridid] = 0;
	}
	if(gridid < numGridsYOv)
	{
		if(topOfBucket3[gridid]!=-1)
		{
			numsift = 0;
			for(i = topOfBucket3[gridid]; gridID3[i] == gridid; i++)
				numsift++;
			numSift3[gridid] = numsift;
		}
		else
			numSift3[gridid] = 0;
	}
	if(gridid < numGridsXYOv)
	{
		if(topOfBucket4[gridid]!=-1)
		{
			numsift = 0;
			for(i = topOfBucket4[gridid]; gridID4[i] == gridid; i++)
				numsift++;
			numSift4[gridid] = numsift;
		}
		else
			numSift4[gridid] = 0;
	}
	__syncthreads();

}


__global__ void findEpipolarLine(float *fmatrix,SiftGPU::SiftKeypoint *keys,int source_width, int source_height,int width,int height,long long int *epipolarPoints,int source_num_keys)
{
	//============  Find Epipolar Line for each source sift ============

	int source_siftID = blockIdx.x*blockDim.x + threadIdx.x;	

	if(source_siftID < source_num_keys)
	{
		float epipolarLine[3];
		short x1,y1,x2,y2;		  // border points of line segment inside the target image
		float top_x, right_y, bottom_x, left_y ;    // points of intersection with the top , right , bottom and left lines of the target image border rectangle
		float x,y;

		// To Euclidian Coordinate system
		x = keys[source_siftID].x - ((source_width-1)/2);
		y = ((source_height-1)/2) - keys[source_siftID].y;

		// E = F.x
		epipolarLine[0] = (fmatrix[0]*x) + (fmatrix[1]*y) + fmatrix[2];
		epipolarLine[1] = (fmatrix[3]*x) + (fmatrix[4]*y) + fmatrix[5];
		epipolarLine[2] = (fmatrix[6]*x) + (fmatrix[7]*y) + fmatrix[8];
		//=====================================================================
		//
		//    In Euclidian Coordinate system (centrailized),
		//    line equations for target image rectangle: 
		//  ---------------------------------------------------------
		//    y=(h-1)/2    (top)
		//    x=(w-1)/2    (right)
		//    y=-(h-1)/2   (bottom)
		//    x=-(w-1)/2   (left)
		//
		//  ---------------------------------------------------------
		//
		//    So points of intersection with epipolar line ( ax + by + c = 0 ) are :
		//  ---------------------------------------------------------
		//    x = (-b(h-1)/2 -c )/a, y = (h-1)/2
		//    x = (w-1)/2, y = (-c-(w-1)a/2)/b
		//    x = (-c+(h-1)b/2)/a , y = -(h-1)/2
		//    x = -(w-1)/2 , y =( -c+(w-1)a/2)/b
		//    
		//=====================================================================

		//================== Find points of intersection ======================

		top_x = ((-1*epipolarLine[1]*(height-1)/2)-(1*epipolarLine[2]))/epipolarLine[0] ;
		right_y =((-1*epipolarLine[2])-((width-1)*epipolarLine[0]/2))/epipolarLine[1];
		bottom_x =((-1*epipolarLine[2])+((height-1)*epipolarLine[1]/2))/epipolarLine[0];
		left_y =((-1*epipolarLine[2])+((width-1)*epipolarLine[0]/2))/epipolarLine[1];

		//=====================================================================

		// Now these points (top , bottom ,left, right) are in Euclidian Coordinate
		// system ((0,0) at the centre of the image)
		// Back to graphics coordinate system ((0,0) at the top left corner of 
		// image and downwards is +ve y)

		top_x = top_x + ((width-1)/2);
		bottom_x = bottom_x + ((width-1)/2);
		right_y = ((height-1)/2) - right_y;
		left_y = ((height-1)/2) - left_y;

		//=====================================================================

		//=====================================================================
		// 
		//   -> Now only two of these points will lie on the target image border rectangle
		//   -> They are (x1,y1) and (x2,y2)
		//   -> Note : x1 < x2
		//
		//=====================================================================

		x1 = x2 = y1 = y2 = 0;
		//====================== Find x1, x2, y1, y2 ==========================

		if(left_y >= 0 && left_y <= height-1)
		{
			if(top_x >= 0 && top_x <= width-1)
			{
				x1 = 0;
				y1 = (short)left_y;
				x2 = (short)top_x;
				y2 = 0;
			}
			else if(bottom_x >= 0 && bottom_x <= width-1)
			{
				x1 = 0;
				y1 = (short)left_y;
				x2 = (short)bottom_x;
				y2 = height-1;
			}
			else if(right_y >= 0 && right_y <= height-1)
			{
				x1 = 0;
				y1 = (short)left_y;
				x2 = width-1;
				y2 = (short)right_y;
			}
		}
		else if(top_x >= 0 && top_x <= width-1)
		{
			if(right_y >= 0 && right_y <= height-1)
			{
				x1 = (short)top_x;
				y1 = 0;
				x2 = width-1;
				y2 = (short)right_y;
			}
			else if(bottom_x >= 0 && bottom_x <= width-1)
			{
				if(top_x < bottom_x)
				{
					x1 = (short)top_x;
					y1 = 0;
					x2 = (short)bottom_x;
					y2 = height-1;

				}
				else
				{
					x2 = (short)top_x;
					y2 = height-1;
					x1 = (short)bottom_x;
					y1 = 0;

				}
			}
		}
		else if(bottom_x >= 0 && bottom_x <= width-1)
		{
			if(right_y >= 0 && right_y <= height-1)
			{
				x1 = (short)bottom_x;
				y1 = height-1;
				x2 = width-1;
				y2 = (short)right_y;
			}

		}

		//=====================================================================

		//======= Packing all these points in a single long long int ==========

		long long int l = 0;
		l = l | (long long int)x1 << 48;
		l = l | (long long int)y1 << 32;
		l = l | (long long int)x2 << 16;
		l = l | (long long int)y2 << 0;
		epipolarPoints[source_siftID] = l;

		//=====================================================================
	}
	__syncthreads();
	// Done !
}


__global__ void clusterLines(long long int *d_epipolarPoints,int *cluster,int source_num_keys)
{

	//=====================================================================
	//
	//   d_epipolarPoints is a sorted array of packed points through which 
	//   we extract the end points of an epipolar line. If there is a 
	//   significant difference between two adjacent end points (say previous 
	//   and current), we mark the current one as 1 else 0 and store it in 
	//   cluster[current]. By default cluster[0] = 0. In this way all the 
	//   points between 1 to the next 1 fall in one group 
	//
	//=====================================================================

	int id = blockIdx.x*blockDim.x + threadIdx.x;	
	// id = 0 : first entry handled on the cpu cluster[0] = 0
	if( id < source_num_keys && id > 0)
	{
		unsigned long long int s = 0,l;
		short x1,y1,x2,y2;
		short prev_x1,prev_y1,prev_x2,prev_y2;
		s = ~s;
		s = s >> 48;

		// Extract the points from the packed point 
		l = d_epipolarPoints[id];
		x1 = (short)(s & (l >> 48));
		y1 = (short)(s & (l >> 32));
		x2 = (short)(s & (l >> 16));
		y2 = (short)(s & (l));

		// Extract the points from the packed point 
		l = d_epipolarPoints[id-1];
		prev_x1 = (short)(s & (l >> 48));
		prev_y1 = (short)(s & (l >> 32));
		prev_x2 = (short)(s & (l >> 16));
		prev_y2 = (short)(s & (l));

		if(((x1-prev_x1)*(x1-prev_x1) < 4) && ((y1-prev_y1)*(y1-prev_y1)< 4) && ((x2-prev_x2)*(x2-prev_x2) < 4) && ((y2-prev_y2)*(y2-prev_y2) < 4))  // If no significant difference between current and previous
		{
			cluster[id] = 0;

		}
		else
		{
			cluster[id] = 1;
		}
	}
	__syncthreads();
	// Done !

}


__global__ void findClusterLocation(int *cluster, int source_num_keys,int *clusterLocation,int *source_siftID,int *clusterID)
{
	//===================================================================
	//
	// Current scene example
	// +-----------+---+---+---+---+---+---+---+---+-----
	// | tidx      | 0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 | ...
	// +-----------+---+---+---+---+---+---+---+---+-----
	// | siftid    | a | b | c | d | e | f | g | h |...
	// +-----------+---+---+---+---+---+---+---+---+-----
	// | cluster   | 0 | 1 | 1 | 2 | 3 | 3 | 3 | 4 |...
	// +-----------+---+---+---+---+---+---+---+---+-----
	//
	// So we want cluster location to be like this so that it 
	// is easy to find out members of one cluster
	// +------------------+---+---+---+---+---+--------
	// | cluster_id       | 0 | 1 | 2 | 3 | 4 | ...
	// +------------------+---+---+---+---+---+--------
	// | cluster_location | 0 | 1 | 3 | 4 | 7 | ...
	// +------------------+---+---+---+---+---+--------
	//
	//===================================================================


	// tidx = 0 handled on the CPU
	int tidx = blockIdx.x*blockDim.x + threadIdx.x;	
	if(tidx < source_num_keys)
	{
		if( tidx != 0) 
		{
			if (cluster[tidx] != cluster[tidx-1])
				clusterLocation[cluster[tidx]] = tidx;
		}
		clusterID[source_siftID[tidx]] = cluster[tidx];
	}
	__syncthreads();

}


__device__ void getBestGrid(float x, float y, int gridSize,int halfSize,int numXGrids1,int numXGrids2,int numGridsXOv,int numGridsYOv,int *gridid, int *which,int numGrids)
{
	int idx[4] = {0,0,0,0};
	float dists[4] = {0,0,0,0};


	double simpleX = ((double)x/gridSize);
	double simpleY = ((double)y/gridSize);


	double ov = (double)halfSize/gridSize;
	idx[0] = floor(simpleY)*numXGrids1 + floor(simpleX);
	idx[1] = floor(simpleY)*numXGrids2 + floor(simpleX - ov);
	idx[2] = floor(simpleY - ov)*numXGrids1 + floor(simpleX);
	idx[3] = floor(simpleY -ov)*numXGrids2 + floor(simpleX - ov);

	if(idx[1] < 0)
		idx[1] = 0;
	if(idx[2] < 0)
		idx[2] = 0;
	if(idx[3] < 0)
		idx[3] = 0;

	float g1Xc = (floor(simpleX))*gridSize + (float)halfSize;
	float g1Yc = (floor(simpleY))*gridSize + (float)halfSize;

	float g2Xc = (floor(simpleX - ov))*gridSize +(float) 2*halfSize;
	//float g2Yc = g1Yc;

	//float g3Xc = g1Xc;
	float g3Yc = (floor(simpleY - ov))*gridSize + (float)2*halfSize;

	//float g4Xc = g2Xc;
	//float g4Yc = g3Yc;

	float x1_d = (g1Xc - x)*(g1Xc - x); 
	float y1_d = (g1Yc - y)*(g1Yc - y);

	float x2_d = (g2Xc - x)*(g2Xc - x);
	float y2_d = (g3Yc - y)*(g3Yc - y);

	dists[0] = x1_d + y1_d;
	dists[1] = x2_d + y1_d;
	dists[2] = x1_d + y2_d;
	dists[3] = x2_d + y2_d;

	float min_g_dist = 200000;
	int minIdx = -1;
	int idno = 0;
	for(int id=0; id < 4; id++) {
		if(dists[id] < min_g_dist && idx[id] >= 0) {
			idno = id+1;
			min_g_dist = dists[id];
			minIdx = idx[id];
		}
	}

	*gridid = minIdx;
	*which = idno;
	//	*gridid = idx[0];
	//	*which = 1;


}




__global__ void	findNumPotentialMatchesForEachCluster(int *gridID,int *target_siftID,int *topOfBucket,int *gridID2,int *target_siftID2,int *topOfBucket2,int *gridID3,int *target_siftID3,int *topOfBucket3,int *gridID4,int *target_siftID4,int *topOfBucket4,int gridSize,int *clusterLocation,long long int *epipolarPoints,int *numSift,int *numSift2,int *numSift3,int *numSift4,int target_height,int numXGrids1,int halfSize,int numXGrids2,int numGridsXOv,int numGridsYOv,int numGrids,int numGridsXYOv,int *numPotentialMatches)	
{
	//=====================================================================
	//
	// For each cluster, load the epipolar line and mark eqidistant points
	// on that line. Now find the respective grids of these points.
	// After knowing which all grids are a part of the epipolar line, The 
	// potential matches are just the target sifts in those grids.
	//
	//=====================================================================

	int clusterID = blockIdx.x;
	int threadID = threadIdx.x;
	int clusterlocation = clusterLocation[clusterID];
	//potentialMatches[maxNumPotentialMatches*clusterID] = 0; 
	extern __shared__ int shared_array[];  

	int i,j,k;
	int distance, d, num_of_points, num_of_pts_per_thread;

	float cos_theta,sin_theta,x,y,slope;

	unsigned long long int s = 0,l;
	short x1,y1,x2,y2;

	s = ~s;
	s = s >> 48;
	l = epipolarPoints[clusterlocation];

	// Extract the points from the packed point l
	x1 = (short)(s & (l >> 48));
	y1 = (short)(s & (l >> 32));
	x2 = (short)(s & (l >> 16));
	y2 = (short)(s & (l));


	distance = (int) sqrt((float)((x1-x2)*(x1-x2)) + ((y1-y2)*(y1-y2)));
	d = gridSize ; 	// equidistant points at d distance
	num_of_points = ceil((float)distance/gridSize);

	if(distance > 5000)
	{
		num_of_points = (5000/gridSize);
		d = distance/num_of_points;
	}

	num_of_pts_per_thread = ceil((float)num_of_points/blockDim.x);


	if((x2-x1) != 0)	// slope is not infinity
	{
		slope = (float)(y2-y1)/(x2-x1);
		cos_theta = 1/sqrt(1+(slope*slope));
		sin_theta = slope/sqrt(1+(slope*slope));
	}
	else			// slope is infinite : vertical epipolar line
	{
		cos_theta = 0;
		sin_theta = 1;
	}

	// Compute gridIDs for num_of_pts_per_thread 

	//==========================================================================
	//
	//   Size of the following array = 10
	//   Reason : I assume that num_of_pts_per_thread can be 10 at max 
	//   if so then num_of_pts = 256(block_size)x10 = 2560
	//   assumption : min gridSize = 2 (4 actually :P )
	//   so length = 2x2560 = 5120 (we have already assumed max length to be 5000)
	//   therefore no problem with this assumption :D
	//  
	//==========================================================================

	int totalNumsift,top;
	int grids[10] = {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1};
	int which[10] = {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1};
	//int gridx,gridy,grid_id;
	int grid_id;

	totalNumsift = 0;
	top = -1;
	
	int which_grid = 0;
	for(i = 0 ; i < num_of_pts_per_thread ; i++ )
	{
		// threadID*num_of_pts_per_thread th point
		x = (float)(x1+(((threadID*num_of_pts_per_thread)+i)*cos_theta*d));
		y = (float)(y1+(((threadID*num_of_pts_per_thread)+i)*sin_theta*d));

		if(top == 10)
			break;

		if( x >= (float)x1 && x <= (float)x2 && y <= (float)target_height-1 && y >= 0.0)	
			// the point lies in the rectangle
		{
			// get best grid id

			// Compute gridID of the point
			getBestGrid(x, y,gridSize, halfSize,numXGrids1, numXGrids2, numGridsXOv,numGridsYOv,&grid_id,&which_grid,numGrids);
			 
			if(top == -1)
			{
				if(which_grid == 1)
				{
					totalNumsift += numSift[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 2)
				{
					totalNumsift += numSift2[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 3)
				{
					totalNumsift += numSift3[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 4)
				{
					totalNumsift += numSift4[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
			}
			else if(grid_id!=grids[top])
			{
				if(which_grid == 1)
				{
					totalNumsift += numSift[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 2)
				{
					totalNumsift += numSift2[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 3)
				{
					totalNumsift += numSift3[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 4)
				{
					totalNumsift += numSift4[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
			}
		}
	}

	shared_array[threadID] = totalNumsift;

	__syncthreads();	
	
	//=========================================================================
	// Find the number of target sifts each thread has to process
	// and write it into shared memory
	// to make sure that all the threads have written their num of sifts in the shared memory
	// Now we need to compute the prefix sum of this shared array which has 256 elements as of now
	// to know the position to write in the global memory for each thread
	//=========================== Prefix Sum ====================================

	//================================ Upsweep ==================================

	if(threadID < 128)
		shared_array[256+threadID] = shared_array[2*threadID]+shared_array[(2*threadID)+1];
	__syncthreads();

	if(threadID < 64)
		shared_array[256+128+threadID] = shared_array[256+(2*threadID)]+shared_array[256+(2*threadID)+1];
	__syncthreads();

	if(threadID < 32)
		shared_array[256+128+64+threadID] = shared_array[256+128+(2*threadID)]+shared_array[256+128+(2*threadID)+1];
	__syncthreads();

	if(threadID < 16)
		shared_array[256+128+64+32+threadID] = shared_array[256+128+64+(2*threadID)]+shared_array[256+128+64+(2*threadID)+1];
	__syncthreads();

	if(threadID < 8)
		shared_array[256+128+64+32+16+threadID] = shared_array[256+128+64+32+(2*threadID)]+shared_array[256+128+64+32+(2*threadID)+1];
	__syncthreads();

	if(threadID < 4)
		shared_array[256+128+64+32+16+8+threadID] = shared_array[256+128+64+32+16+(2*threadID)]+shared_array[256+128+64+32+16+(2*threadID)+1];
	__syncthreads();

	if(threadID < 2)
		shared_array[256+128+64+32+16+8+4+threadID] = shared_array[256+128+64+32+16+8+(2*threadID)]+shared_array[256+128+64+32+16+8+(2*threadID)+1];
	__syncthreads();

	if(threadID < 1)
		shared_array[256+128+64+32+16+8+4+2+threadID] = shared_array[256+128+64+32+16+8+4+(2*threadID)]+shared_array[256+128+64+32+16+8+4+(2*threadID)+1];
	__syncthreads();


	//======================== Downsweep ==============================

	if(threadID < 2 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+64+32+16+8+4+threadID] = shared_array[256+128+64+32+16+8+4+2+(threadID/2)];
		else
			shared_array[256+128+64+32+16+8+4+threadID] += shared_array[256+128+64+32+16+8+4+2+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 4 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+64+32+16+8+threadID] = shared_array[256+128+64+32+16+8+4+(threadID/2)];
		else
			shared_array[256+128+64+32+16+8+threadID] += shared_array[256+128+64+32+16+8+4+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 8 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+64+32+16+threadID] = shared_array[256+128+64+32+16+8+(threadID/2)];
		else
			shared_array[256+128+64+32+16+threadID] += shared_array[256+128+64+32+16+8+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 16 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+64+32+threadID] = shared_array[256+128+64+32+16+(threadID/2)];
		else
			shared_array[256+128+64+32+threadID] += shared_array[256+128+64+32+16+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 32 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+64+threadID] = shared_array[256+128+64+32+(threadID/2)];
		else
			shared_array[256+128+64+threadID] += shared_array[256+128+64+32+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 64 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+threadID] = shared_array[256+128+64+(threadID/2)];
		else
			shared_array[256+128+threadID] += shared_array[256+128+64+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 128 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+threadID] = shared_array[256+128+(threadID/2)];
		else
			shared_array[256+threadID] += shared_array[256+128+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 256 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[threadID] = shared_array[256+(threadID/2)];
		else
			shared_array[threadID] += shared_array[256+(threadID/2)-1];
	}
	__syncthreads();

	//==========================================================================
	//	
	// Now shared array[0..255] contains prefix sum
	// There is an array of size maxNumPotentialMatches for each cluster, we will write the potential matches there
	// The 1st entry would be the total number of potential matches and 
	// then each thread will write the matching sift ids it found at write position
	//
	//==========================================================================


	if(threadID == 0)
	{
//		printf("%d \n",shared_array[255]);
		numPotentialMatches[clusterID] = shared_array[255];
	}

	__syncthreads();

}



__global__ void	findPotentialMatchesForEachCluster(int *gridID,int *target_siftID,int *topOfBucket,int *gridID2,int *target_siftID2,int *topOfBucket2,int *gridID3,int *target_siftID3,int *topOfBucket3,int *gridID4,int *target_siftID4,int *topOfBucket4,int gridSize,int *clusterLocation,long long int *epipolarPoints,int *numSift,int *numSift2,int *numSift3,int *numSift4,int *potentialMatches,int target_height,int numXGrids1,int halfSize,int numXGrids2,int numGridsXOv,int numGridsYOv,int numGrids,int numGridsXYOv,int maxNumPotentialMatches)	
{
	//=====================================================================
	//
	// For each cluster, load the epipolar line and mark eqidistant points
	// on that line. Now find the respective grids of these points.
	// After knowing which all grids are a part of the epipolar line, The 
	// potential matches are just the target sifts in those grids.
	//
	//=====================================================================

	int clusterID = blockIdx.x;
	int threadID = threadIdx.x;
	int clusterlocation = clusterLocation[clusterID];
	potentialMatches[(2+maxNumPotentialMatches)*clusterID] = 0; 
	extern __shared__ int shared_array[];  

	int i,j,k;
	int distance, d, num_of_points, num_of_pts_per_thread;

	float cos_theta,sin_theta,x,y,slope;

	unsigned long long int s = 0,l;
	short x1,y1,x2,y2;

	s = ~s;
	s = s >> 48;
	l = epipolarPoints[clusterlocation];

	// Extract the points from the packed point l
	x1 = (short)(s & (l >> 48));
	y1 = (short)(s & (l >> 32));
	x2 = (short)(s & (l >> 16));
	y2 = (short)(s & (l));


	distance = (int) sqrt((float)((x1-x2)*(x1-x2)) + ((y1-y2)*(y1-y2)));
	d = gridSize ; 	// equidistant points at d distance
	num_of_points = ceil((float)distance/gridSize);

	if(distance > 5000)
	{
		num_of_points = (5000/gridSize);
		d = distance/num_of_points;
	}

	num_of_pts_per_thread = ceil((float)num_of_points/blockDim.x);


	if((x2-x1) != 0)	// slope is not infinity
	{
		slope = (float)(y2-y1)/(x2-x1);
		cos_theta = 1/sqrt(1+(slope*slope));
		sin_theta = slope/sqrt(1+(slope*slope));
	}
	else			// slope is infinite : vertical epipolar line
	{
		cos_theta = 0;
		sin_theta = 1;
	}

	// Compute gridIDs for num_of_pts_per_thread 

	//==========================================================================
	//
	//   Size of the following array = 10
	//   Reason : I assume that num_of_pts_per_thread can be 10 at max 
	//   if so then num_of_pts = 256(block_size)x10 = 2560
	//   assumption : min gridSize = 2 (4 actually :P )
	//   so length = 2x2560 = 5120 (we have already assumed max length to be 5000)
	//   therefore no problem with this assumption :D
	//  
	//==========================================================================

	int totalNumsift,top;
	int grids[10] = {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1};
	int which[10] = {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1};
	//int gridx,gridy,grid_id;
	int grid_id;

	totalNumsift = 0;
	top = -1;
	
	int which_grid = 0;
	for(i = 0 ; i < num_of_pts_per_thread ; i++ )
	{
		// threadID*num_of_pts_per_thread th point
		x = (float)(x1+(((threadID*num_of_pts_per_thread)+i)*cos_theta*d));
		y = (float)(y1+(((threadID*num_of_pts_per_thread)+i)*sin_theta*d));
		if(top == 10)
			break;

		if( x >= (float)x1 && x <= (float)x2 && y <= (float)target_height-1 && y >= 0.0)	
			// the point lies in the rectangle
		{
			// get best grid id

			// Compute gridID of the point
			getBestGrid(x, y,gridSize, halfSize,numXGrids1, numXGrids2, numGridsXOv,numGridsYOv,&grid_id,&which_grid,numGrids);
			 
			if(top == -1)
			{
				if(which_grid == 1)
				{
					totalNumsift += numSift[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 2)
				{
					totalNumsift += numSift2[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 3)
				{
					totalNumsift += numSift3[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 4)
				{
					totalNumsift += numSift4[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
			}
			else if(grid_id!=grids[top])
			{
				if(which_grid == 1)
				{
					totalNumsift += numSift[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 2)
				{
					totalNumsift += numSift2[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 3)
				{
					totalNumsift += numSift3[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
				else if(which_grid == 4)
				{
					totalNumsift += numSift4[grid_id];
					grids[++top] = grid_id;
					which[top] = which_grid;
				}
			}
		}
	}

	shared_array[threadID] = totalNumsift;

	__syncthreads();	
	
	//=========================================================================
	// Find the number of target sifts each thread has to process
	// and write it into shared memory
	// to make sure that all the threads have written their num of sifts in the shared memory
	// Now we need to compute the prefix sum of this shared array which has 256 elements as of now
	// to know the position to write in the global memory for each thread
	//=========================== Prefix Sum ====================================

	//================================ Upsweep ==================================

	if(threadID < 128)
		shared_array[256+threadID] = shared_array[2*threadID]+shared_array[(2*threadID)+1];
	__syncthreads();

	if(threadID < 64)
		shared_array[256+128+threadID] = shared_array[256+(2*threadID)]+shared_array[256+(2*threadID)+1];
	__syncthreads();

	if(threadID < 32)
		shared_array[256+128+64+threadID] = shared_array[256+128+(2*threadID)]+shared_array[256+128+(2*threadID)+1];
	__syncthreads();

	if(threadID < 16)
		shared_array[256+128+64+32+threadID] = shared_array[256+128+64+(2*threadID)]+shared_array[256+128+64+(2*threadID)+1];
	__syncthreads();

	if(threadID < 8)
		shared_array[256+128+64+32+16+threadID] = shared_array[256+128+64+32+(2*threadID)]+shared_array[256+128+64+32+(2*threadID)+1];
	__syncthreads();

	if(threadID < 4)
		shared_array[256+128+64+32+16+8+threadID] = shared_array[256+128+64+32+16+(2*threadID)]+shared_array[256+128+64+32+16+(2*threadID)+1];
	__syncthreads();

	if(threadID < 2)
		shared_array[256+128+64+32+16+8+4+threadID] = shared_array[256+128+64+32+16+8+(2*threadID)]+shared_array[256+128+64+32+16+8+(2*threadID)+1];
	__syncthreads();

	if(threadID < 1)
		shared_array[256+128+64+32+16+8+4+2+threadID] = shared_array[256+128+64+32+16+8+4+(2*threadID)]+shared_array[256+128+64+32+16+8+4+(2*threadID)+1];
	__syncthreads();


	//======================== Downsweep ==============================

	if(threadID < 2 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+64+32+16+8+4+threadID] = shared_array[256+128+64+32+16+8+4+2+(threadID/2)];
		else
			shared_array[256+128+64+32+16+8+4+threadID] += shared_array[256+128+64+32+16+8+4+2+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 4 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+64+32+16+8+threadID] = shared_array[256+128+64+32+16+8+4+(threadID/2)];
		else
			shared_array[256+128+64+32+16+8+threadID] += shared_array[256+128+64+32+16+8+4+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 8 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+64+32+16+threadID] = shared_array[256+128+64+32+16+8+(threadID/2)];
		else
			shared_array[256+128+64+32+16+threadID] += shared_array[256+128+64+32+16+8+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 16 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+64+32+threadID] = shared_array[256+128+64+32+16+(threadID/2)];
		else
			shared_array[256+128+64+32+threadID] += shared_array[256+128+64+32+16+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 32 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+64+threadID] = shared_array[256+128+64+32+(threadID/2)];
		else
			shared_array[256+128+64+threadID] += shared_array[256+128+64+32+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 64 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+128+threadID] = shared_array[256+128+64+(threadID/2)];
		else
			shared_array[256+128+threadID] += shared_array[256+128+64+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 128 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[256+threadID] = shared_array[256+128+(threadID/2)];
		else
			shared_array[256+threadID] += shared_array[256+128+(threadID/2)-1];
	}
	__syncthreads();

	if(threadID < 256 && threadID!=0)	
	{
		if(threadID%2 != 0)
			shared_array[threadID] = shared_array[256+(threadID/2)];
		else
			shared_array[threadID] += shared_array[256+(threadID/2)-1];
	}
	__syncthreads();


	//==========================================================================
	//	
	// Now shared array[0..255] contains prefix sum
	// There is an array of size maxNumPotentialMatches for each cluster, we will write the potential matches there
	// The 1st entry would be the total number of potential matches and 
	// then each thread will write the matching sift ids it found at write position
	//
	//==========================================================================

	potentialMatches[(2+maxNumPotentialMatches)*clusterID] = shared_array[255];	// total num of potential matches

	int writePosition = 1;

	if(threadID > 0)
		writePosition = shared_array[threadID-1] + 1;

	__syncthreads();



	k = 0;

	for(i=0;i<=top;i++)
	{
		if(which[i] == 1)
		{
			if(topOfBucket[grids[i]]!=-1)
			{
				for(j = topOfBucket[grids[i]]; grids[i] == gridID[j]; j++)
				{
					potentialMatches[((2+maxNumPotentialMatches)*clusterID)+writePosition+k] = target_siftID[j];
					k++;
				}

			}
		}
		else if(which[i] == 2)
		{
			if(topOfBucket2[grids[i]]!=-1)
			{
				for(j = topOfBucket2[grids[i]]; grids[i] == gridID2[j]; j++)
				{
					potentialMatches[((2+maxNumPotentialMatches)*clusterID)+writePosition+k] = target_siftID2[j];
					k++;
				}

			}
		}
		else if(which[i] == 3)
		{
			if(topOfBucket3[grids[i]]!=-1)
			{
				for(j = topOfBucket3[grids[i]]; grids[i] == gridID3[j]; j++)
				{
					potentialMatches[((2+maxNumPotentialMatches)*clusterID)+writePosition+k] = target_siftID3[j];
					k++;
				}

			}
		}
		else if(which[i] == 4)
		{
			if(topOfBucket4[grids[i]]!=-1)
			{
				for(j = topOfBucket4[grids[i]]; grids[i] == gridID4[j]; j++)
				{
					potentialMatches[((2+maxNumPotentialMatches)*clusterID)+writePosition+k] = target_siftID4[j];
					k++;
				}

			}
		}
	}
	__syncthreads();

}



__device__ void findmin1min2(int a, int b , int c, int d,int a_id,int b_id,int c_id,int d_id,int *min1,int *min2,int *min1_id,int *min2_id)
{
	//========================================================================
	// This chunk of code finds the minimum and next minimun and correnponding 
	// minimum and next minimum ID between a,b,c,d 
	//========================================================================

	if( a < b && a < c && a < d )
	{
		*min1 = a;
		*min1_id = a_id;
		if(b < c && b < d)
		{
			*min2 = b;
			*min2_id = b_id;
		}
		else if(c < d)
		{
			*min2 = c;
			*min2_id = c_id;
		}
		else
		{
			*min2 = d;
			*min2_id = d_id;
		}
	}
	else if(b < c && b < d)
	{
		*min1 = b;
		*min1_id = b_id;
		if(a < c && a < d)
		{
			*min2 = a;
			*min2_id = a_id;
		}
		else if(c < d)
		{
			*min2 = c;
			*min2_id = c_id;
		}
		else
		{
			*min2 = d;
			*min2_id = d_id;
		}
	}
	else if(c < d)
	{
		*min1 = c;
		*min1_id = c_id;
		if(a < b && a < d)
		{
			*min2 = a;
			*min2_id = a_id;
		}
		else if(b < d)
		{
			*min2 = b;
			*min2_id = b_id;
		}
		else
		{
			*min2 = d;
			*min2_id = d_id;
		}
	}
	else
	{
		*min1 = d;
		*min1_id = d_id;
		if(a < b && a < c)
		{
			*min2 = a;
			*min2_id = a_id;
		}
		else if(b < c)
		{
			*min2 = b;
			*min2_id = b_id;
		}
		else
		{
			*min2 = c;
			*min2_id = c_id;
		}
	}


}



__global__ void	findMatches(int *matches,int *clusterID,int *potentialMatches,int maxNumPotentialMatches)
{
	//========================================================================
	//
	// For each source sift, find the cluster it belongs to 
	// Then load the potential matching target sifts of that cluster and 
	// compute the L2 distance between the descriptors.
	// Now find the min and next min of these distances.
	// Declare a match based on some tests.
	//
	//========================================================================

	int source_siftID,threadID,cluster_id,num_potential_matches,num_pts_per_thread,target_siftID;
	int min_id,next_min_id,i,j;
	int min_num,next_min,dist;

	extern __shared__ int shared_array[];  

	source_siftID = blockIdx.x;
	threadID = threadIdx.x;
	cluster_id = clusterID[source_siftID];
	num_potential_matches = potentialMatches[(2+maxNumPotentialMatches)*cluster_id];

//	matches[source_siftID] = -1;

	if (num_potential_matches%blockDim.x == 0)
		num_pts_per_thread = num_potential_matches/blockDim.x;
	else
		num_pts_per_thread = 1 + (num_potential_matches/blockDim.x);


//	if(num_potential_matches > maxNumPotentialMatches)
//		printf("%d\n",num_potential_matches);

	min_num = 999999;
	next_min = 999999;
	min_id = -1;
	next_min_id = -1;
	int dummy;
	int a,b;

	for(i = 0;i < num_pts_per_thread; i++)
	{
		if((num_pts_per_thread*threadID)+i < num_potential_matches)
		{
			target_siftID = potentialMatches[((2+maxNumPotentialMatches)*cluster_id)+1+(num_pts_per_thread*threadID)+i];
			dist = 0;
			for(j = 0 ; j < 128; j++)
			{
				a = tex1Dfetch(tex_targetDescriptor,(target_siftID*128)+j);
				b = tex1Dfetch(tex_sourceDescriptor,(source_siftID*128)+j);
				dummy = a-b;
				dist+=(dummy*dummy);

			}

	
//			for( j = 0 ; j < 128 ; j++) 
//			{
//				dist+= (((int)tex1Dfetch(tex_sourceDescriptor,(source_siftID*128)+j) - ((int)tex1Dfetch(tex_targetDescriptor,(target_siftID*128)+j)))*((int)tex1Dfetch(tex_sourceDescriptor,(source_siftID*128)+j) - ((int)tex1Dfetch(tex_targetDescriptor,(target_siftID*128)+j))));
//			}

			if(dist <= min_num)
			{
				next_min = min_num;
				next_min_id = min_id;
				min_num = dist;
				min_id = target_siftID;
			}
			else if(dist < next_min)
			{
				next_min = dist;
				next_min_id = target_siftID;
			}
		}
	}

	// Store the min and next min found by each thread into the shared memory

	shared_array[threadID] = min_num;
	shared_array[threadID+blockDim.x] = next_min;
	shared_array[threadID+2*blockDim.x] = min_id;
	shared_array[threadID+3*blockDim.x] = next_min_id;
	__syncthreads();

	// Shared array fully occupied
	// Time to compute the min and next min across the block ( min and next min in the shared memory )

	int c,d;
	int a_id,b_id,c_id,d_id;
	int min1,min2;
	int min1_id,min2_id;

	if(threadID < 128)
	{
		a = shared_array[threadID];
		b = shared_array[threadID+blockDim.x];
		a_id = shared_array[threadID+2*blockDim.x];
		b_id = shared_array[threadID+3*blockDim.x];

		c = shared_array[threadID+128];
		d = shared_array[threadID+blockDim.x+128];
		c_id = shared_array[threadID+2*blockDim.x+128];
		d_id = shared_array[threadID+3*blockDim.x+128];

		findmin1min2(a,b,c,d,a_id,b_id,c_id,d_id,&min1,&min2,&min1_id,&min2_id);

		shared_array[threadID] = min1;
		shared_array[threadID+blockDim.x] = min2;
		shared_array[threadID+2*blockDim.x] = min1_id;
		shared_array[threadID+3*blockDim.x] = min2_id;
	}
	__syncthreads();

	if(threadID < 64)
	{
		a = shared_array[threadID];
		b = shared_array[threadID+blockDim.x];
		a_id = shared_array[threadID+2*blockDim.x];
		b_id = shared_array[threadID+3*blockDim.x];

		c = shared_array[threadID+64];
		d = shared_array[threadID+blockDim.x+64];
		c_id = shared_array[threadID+2*blockDim.x+64];
		d_id = shared_array[threadID+3*blockDim.x+64];

		findmin1min2(a,b,c,d,a_id,b_id,c_id,d_id,&min1,&min2,&min1_id,&min2_id);

		shared_array[threadID] = min1;
		shared_array[threadID+blockDim.x] = min2;
		shared_array[threadID+2*blockDim.x] = min1_id;
		shared_array[threadID+3*blockDim.x] = min2_id;
	}
	__syncthreads();

	if(threadID < 32)
	{
		a = shared_array[threadID];
		b = shared_array[threadID+blockDim.x];
		a_id = shared_array[threadID+2*blockDim.x];
		b_id = shared_array[threadID+3*blockDim.x];

		c = shared_array[threadID+32];
		d = shared_array[threadID+blockDim.x+32];
		c_id = shared_array[threadID+2*blockDim.x+32];
		d_id = shared_array[threadID+3*blockDim.x+32];

		findmin1min2(a,b,c,d,a_id,b_id,c_id,d_id,&min1,&min2,&min1_id,&min2_id);

		shared_array[threadID] = min1;
		shared_array[threadID+blockDim.x] = min2;
		shared_array[threadID+2*blockDim.x] = min1_id;
		shared_array[threadID+3*blockDim.x] = min2_id;
	}
	__syncthreads();

	if(threadID < 16)
	{
		a = shared_array[threadID];
		b = shared_array[threadID+blockDim.x];
		a_id = shared_array[threadID+2*blockDim.x];
		b_id = shared_array[threadID+3*blockDim.x];

		c = shared_array[threadID+16];
		d = shared_array[threadID+blockDim.x+16];
		c_id = shared_array[threadID+2*blockDim.x+16];
		d_id = shared_array[threadID+3*blockDim.x+16];

		findmin1min2(a,b,c,d,a_id,b_id,c_id,d_id,&min1,&min2,&min1_id,&min2_id);

		shared_array[threadID] = min1;
		shared_array[threadID+blockDim.x] = min2;
		shared_array[threadID+2*blockDim.x] = min1_id;
		shared_array[threadID+3*blockDim.x] = min2_id;
	}
	__syncthreads();

	if(threadID < 8)
	{
		a = shared_array[threadID];
		b = shared_array[threadID+blockDim.x];
		a_id = shared_array[threadID+2*blockDim.x];
		b_id = shared_array[threadID+3*blockDim.x];

		c = shared_array[threadID+8];
		d = shared_array[threadID+blockDim.x+8];
		c_id = shared_array[threadID+2*blockDim.x+8];
		d_id = shared_array[threadID+3*blockDim.x+8];

		findmin1min2(a,b,c,d,a_id,b_id,c_id,d_id,&min1,&min2,&min1_id,&min2_id);

		shared_array[threadID] = min1;
		shared_array[threadID+blockDim.x] = min2;
		shared_array[threadID+2*blockDim.x] = min1_id;
		shared_array[threadID+3*blockDim.x] = min2_id;
	}
	__syncthreads();

	if(threadID < 4)
	{
		a = shared_array[threadID];
		b = shared_array[threadID+blockDim.x];
		a_id = shared_array[threadID+2*blockDim.x];
		b_id = shared_array[threadID+3*blockDim.x];

		c = shared_array[threadID+4];
		d = shared_array[threadID+blockDim.x+4];
		c_id = shared_array[threadID+2*blockDim.x+4];
		d_id = shared_array[threadID+3*blockDim.x+4];

		findmin1min2(a,b,c,d,a_id,b_id,c_id,d_id,&min1,&min2,&min1_id,&min2_id);

		shared_array[threadID] = min1;
		shared_array[threadID+blockDim.x] = min2;
		shared_array[threadID+2*blockDim.x] = min1_id;
		shared_array[threadID+3*blockDim.x] = min2_id;
	}
	__syncthreads();

	if(threadID < 2)
	{
		a = shared_array[threadID];
		b = shared_array[threadID+blockDim.x];
		a_id = shared_array[threadID+2*blockDim.x];
		b_id = shared_array[threadID+3*blockDim.x];

		c = shared_array[threadID+2];
		d = shared_array[threadID+blockDim.x+2];
		c_id = shared_array[threadID+2*blockDim.x+2];
		d_id = shared_array[threadID+3*blockDim.x+2];

		findmin1min2(a,b,c,d,a_id,b_id,c_id,d_id,&min1,&min2,&min1_id,&min2_id);

		shared_array[threadID] = min1;
		shared_array[threadID+blockDim.x] = min2;
		shared_array[threadID+2*blockDim.x] = min1_id;
		shared_array[threadID+3*blockDim.x] = min2_id;
	}
	__syncthreads();

	if(threadID == 0)
	{
		a = shared_array[threadID];
		b = shared_array[threadID+blockDim.x];
		a_id = shared_array[threadID+2*blockDim.x];
		b_id = shared_array[threadID+3*blockDim.x];

		c = shared_array[threadID+1];
		d = shared_array[threadID+blockDim.x+1];
		c_id = shared_array[threadID+2*blockDim.x+1];
		d_id = shared_array[threadID+3*blockDim.x+1];

		// Compute min and next min
		findmin1min2(a,b,c,d,a_id,b_id,c_id,d_id,&min1,&min2,&min1_id,&min2_id);

		// Ratio test
		if((float)min1/min2 < 0.36 && min1_id > 0 && num_potential_matches > 9 && min2_id != 999999)	// we store the squared distances so the ratio becomes 0.36 instead of 0.6
		{
			// Declare match
			matches[source_siftID] = min1_id;

		}
	}
	__syncthreads();

}

int max(int a,int b,int c,int d)
{
	int maxim = -9999;
	if(maxim < a)
		maxim = a;
	if(maxim < b)
		maxim = b;
	if(maxim < c)
		maxim = c;
	if(maxim < d)
		maxim = d;
	return maxim;

}


double Matcher::matchImagePair(sift_img source,sift_img target,float *f_matrix,SiftGPU::SiftKeypoint* &d_keys1,unsigned char* &descriptors1 ,SiftGPU::SiftKeypoint* &d_keys2,unsigned char* &descriptors2 )
{

	struct timespec t1, t2, t3, t4, t5;

	clock_gettime(CLOCK_MONOTONIC, &t3);
	clock_gettime(CLOCK_MONOTONIC, &t1);

	//================== GPU initialization =======================

	hipSetDevice(0);
	hipFree(0);

	//=============================================================

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_gpu_init = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//=============================================================

	//=============================================================

	clock_gettime(CLOCK_MONOTONIC, &t1);
	//================== Declare variables ========================


	int i,j;
	int *h_cluster, *h_gridID;
	long long int *h_epipolarPoints; // Stores ends of the epipolar line segment packed in a single long long int

	h_epipolarPoints = (long long int*) malloc(sizeof(long long int)*source.num_keys); 
	h_cluster = (int *) malloc(sizeof(int)*source.num_keys);
	h_gridID = (int *) malloc(sizeof(int)*target.num_keys);	//to store gridIDs on host

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_total_cpu_alloc = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	clock_gettime(CLOCK_MONOTONIC, &t1);
	//==================== Declare device arrays ===================

	clock_gettime(CLOCK_MONOTONIC, &t5);
//	float *d_target_x, *d_target_y, *d_source_x, *d_source_y;
//	SiftGPU::SiftKeypoint *d_keys1, *d_keys2;


	float *d_fmatrix;
//	float *desc1,*desc2;
	int *d_gridID , *d_target_siftID , *d_gridID1 , *d_target_siftID1, *d_gridID2 , *d_target_siftID2,*d_gridID3 , *d_target_siftID3,*d_gridID4 , *d_target_siftID4,   *d_source_siftID, *d_cluster, *d_clusterID;
////	unsigned char *d_source_keypoints,*d_target_keypoints;
	long long int *d_epipolarPoints;

	//==============================================================

	//=================== Allocate memory on device ================

//	hipMalloc((void**)&d_keys1, sizeof(SiftGPU::SiftKeypoint)*source.num_keys);
//	hipMalloc((void**)&d_keys2, sizeof(SiftGPU::SiftKeypoint)*target.num_keys);
//	hipMalloc((void**)&desc1, 128*sizeof(float)*source.num_keys);
//	hipMalloc((void**)&desc2, 128*sizeof(float)*target.num_keys);
//	hipMalloc((void**)&d_source_x, sizeof(float)*source.num_keys);
//	hipMalloc((void**)&d_source_y, sizeof(float)*source.num_keys);

//	hipMalloc((void**)&d_target_x, sizeof(float)*target.num_keys);
//	hipMalloc((void**)&d_target_y, sizeof(float)*target.num_keys);

////	hipMalloc((void**)&d_source_keypoints, sizeof(unsigned char)*128*source.num_keys);
////	hipMalloc((void**)&d_target_keypoints, sizeof(unsigned char)*128*target.num_keys);

	hipMalloc((void**)&d_source_siftID, sizeof(int)*source.num_keys);
	hipMalloc((void**)&d_target_siftID, sizeof(int)*target.num_keys);
	hipMalloc((void**)&d_target_siftID1, sizeof(int)*target.num_keys);
	hipMalloc((void**)&d_target_siftID2, sizeof(int)*target.num_keys);
	hipMalloc((void**)&d_target_siftID3, sizeof(int)*target.num_keys);
	hipMalloc((void**)&d_target_siftID4, sizeof(int)*target.num_keys);

	hipMalloc((void**)&d_fmatrix, sizeof(float)*9);

	hipMalloc((void**)&d_gridID, sizeof(int)*target.num_keys);
	hipMalloc((void**)&d_gridID1, sizeof(int)*target.num_keys);
	hipMalloc((void**)&d_gridID2, sizeof(int)*target.num_keys);
	hipMalloc((void**)&d_gridID3, sizeof(int)*target.num_keys);
	hipMalloc((void**)&d_gridID4, sizeof(int)*target.num_keys);

	hipMalloc((void**)&d_epipolarPoints, sizeof(long long int)*source.num_keys);

	hipMalloc((void**)&d_cluster, sizeof(int)*source.num_keys);
	hipMalloc((void**)&d_clusterID, sizeof(int)*source.num_keys);


	//==============================================================

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_malloc1 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);
	//================ Copy arrays from host to device =============



//	hipMemcpy(desc1, descriptors1.data(), sizeof(float)*source.num_keys*128, hipMemcpyHostToDevice); 
//	hipMemcpy(desc2, descriptors2.data(), sizeof(float)*target.num_keys*128, hipMemcpyHostToDevice); 
	clock_gettime(CLOCK_MONOTONIC, &t1);

//	hipMemcpy(d_keys1, keys1.data(), sizeof(SiftGPU::SiftKeypoint)*source.num_keys, hipMemcpyHostToDevice); 
//	hipMemcpy(d_keys2, keys2.data(), sizeof(SiftGPU::SiftKeypoint)*target.num_keys, hipMemcpyHostToDevice); 

	hipMemcpy(d_fmatrix, f_matrix, sizeof(float)*9, hipMemcpyHostToDevice); 

//	hipMemcpy(d_target_keypoints, target.keypoints, sizeof(unsigned char)*target.num_keys*128, hipMemcpyHostToDevice); 
//	hipMemcpy(d_source_keypoints, source.keypoints, sizeof(unsigned char)*source.num_keys*128, hipMemcpyHostToDevice); 

	//==============================================================

	int threadsPerBlock;
	int numBlocks;
	thrust::device_ptr<int> d_target_sift_ptr(d_target_siftID);
	thrust::device_ptr<int> d_source_sift_ptr(d_source_siftID);
	thrust::sequence(d_target_sift_ptr,d_target_sift_ptr+target.num_keys);
	thrust::sequence(d_source_sift_ptr,d_source_sift_ptr+source.num_keys);
	
	// initialize a device_vector with the list
////	thrust::host_vector<unsigned char> h_d1(descriptors1.begin(), descriptors1.end());
////	thrust::host_vector<unsigned char> h_d2(descriptors2.begin(), descriptors2.end());
////	thrust::device_vector<unsigned char> d1 = h_d1;
////	thrust::device_vector<unsigned char> d2 = h_d2; 
//	thrust::device_ptr<unsigned char> d_desc1_ptr(desc1);
//	thrust::device_ptr<unsigned char> d_desc2_ptr(desc2);
//	thrust::device_vector<unsigned char> u_d1(128*source.num_keys);
//	thrust::device_vector<unsigned char> u_d2(128*target.num_keys);
//	thrust::transform(d1.begin(), d1.end(), u_d1.begin(), cast());
//	thrust::transform(d2.begin(), d2.end(), u_d2.begin(), cast());

//	thrust::transform(d_desc1_ptr,d_desc1_ptr + 128*source.num_keys, u_d1.begin(), cast());
//	thrust::transform(d_desc2_ptr,d_desc2_ptr + 128*target.num_keys, u_d2.begin(), cast());

////	unsigned char * desc1_ptr = thrust::raw_pointer_cast(d1.data());
////	unsigned char * desc2_ptr = thrust::raw_pointer_cast(d2.data());

//	unsigned char *h = (unsigned char *)malloc(128*target.num_keys*sizeof(unsigned char));
//	unsigned char * dv_ptr = thrust::raw_pointer_cast(u_d2.data());
//	hipMemcpy(h,dv_ptr,128*sizeof(unsigned char)*target.num_keys, hipMemcpyDeviceToHost);

//	for(int i = 0; i < 128*target.num_keys; i++)
//	{
//		printf("%hhu\n",h[i]);
//	}



	hipMemcpy(d_target_siftID1, d_target_siftID, sizeof(int)*target.num_keys, hipMemcpyDeviceToDevice); 
	hipMemcpy(d_target_siftID2, d_target_siftID, sizeof(int)*target.num_keys, hipMemcpyDeviceToDevice); 
	hipMemcpy(d_target_siftID3, d_target_siftID, sizeof(int)*target.num_keys, hipMemcpyDeviceToDevice); 
	hipMemcpy(d_target_siftID4, d_target_siftID, sizeof(int)*target.num_keys, hipMemcpyDeviceToDevice); 

	//====================== Bind textures =========================

	hipBindTexture (0, tex_targetDescriptor, descriptors2, sizeof(unsigned char)*target.num_keys*128 );
	hipBindTexture (0, tex_sourceDescriptor, descriptors1, sizeof(unsigned char)*source.num_keys*128 );

//	hipBindTexture (0, tex_targetDescriptor, d_target_keypoints, sizeof(unsigned char)*target.num_keys*128 );
//	hipBindTexture (0, tex_sourceDescriptor, d_source_keypoints, sizeof(unsigned char)*source.num_keys*128 );

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_memcpy1 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);
	//==============================================================


	//printf("%d %d\n",source.num_keys,target.num_keys);

	//======================  INDEXING STEP ========================
	// The indexing step divides the target image into grids. 
	// The features get indexed based on the grid to which they belong.
	//==============================================================

	Grid target_grid;
	target_grid.g_init(target);

	//================ Decide kernel configuration =================

	threadsPerBlock = 512;
	numBlocks = ceil(((float)target.num_keys)/threadsPerBlock);

	//===================== Call the GPU Kernel ====================
	// This kernel computes gridID for each sift of the target image

	clock_gettime(CLOCK_MONOTONIC, &t1);

	
	computeGridID<<<numBlocks, threadsPerBlock>>>(d_keys2, d_gridID1, d_gridID2, d_gridID3, d_gridID4, target.num_keys,target_grid.gridSize, target_grid.numXGrids1,target_grid.halfSize,target_grid.numXGrids2,target_grid.numGrids,target_grid.numGridsXOv,target_grid.numGridsYOv);
	
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_computeGridID = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//==============================================================


	// Current situation 
	// +-------+---+---+---+---+---+---+---+----
	// |SiftID | 0 | 1 | 2 | 3 | 4 | 5 | 6 | ...
	// +-------+---+---+---+---+---+---+---+----
	// |GridID | a | b | c | d | e | f | g | ...
	// +-------+---+---+---+---+---+---+---+----
	// Desired situation
	// +-------+---+---+---+---+---+---+---+----
	// |GridID | 0 | 0 | 2 | 5 | 5 | 5 | 6 | ...
	// +-------+---+---+---+---+---+---+---+----
	// |SiftID | a | b | c | d | e | f | g | ...
	// +-------+---+---+---+---+---+---+---+----
	// Sort to bring sifts with same gridID together

	//========= Sort : keys = gridID , values = siftID ===========

	clock_gettime(CLOCK_MONOTONIC, &t1);

	thrust::device_ptr<int> d_gridID_ptr1(d_gridID1);
	thrust::device_ptr<int> d_target_siftID_ptr1(d_target_siftID1);

	thrust::device_ptr<int> d_gridID_ptr2(d_gridID2);
	thrust::device_ptr<int> d_target_siftID_ptr2(d_target_siftID2);

	thrust::device_ptr<int> d_gridID_ptr3(d_gridID3);
	thrust::device_ptr<int> d_target_siftID_ptr3(d_target_siftID3);

	thrust::device_ptr<int> d_gridID_ptr4(d_gridID4);
	thrust::device_ptr<int> d_target_siftID_ptr4(d_target_siftID4);

	thrust::sort_by_key(d_gridID_ptr1, d_gridID_ptr1+target.num_keys,d_target_siftID_ptr1);
	thrust::sort_by_key(d_gridID_ptr2, d_gridID_ptr2+target.num_keys,d_target_siftID_ptr2);
	thrust::sort_by_key(d_gridID_ptr3, d_gridID_ptr3+target.num_keys,d_target_siftID_ptr3);
	thrust::sort_by_key(d_gridID_ptr4, d_gridID_ptr4+target.num_keys,d_target_siftID_ptr4);

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_thrust1 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);


	// Current situation 
	// +-------+---+---+---+---+---+---+---+----
	// |GridID | 0 | 0 | 2 | 5 | 5 | 5 | 6 | ...
	// +-------+---+---+---+---+---+---+---+----
	// |SiftID | a | b | c | d | e | f | g | ...
	// +-------+---+---+---+---+---+---+---+----
	// Now to access all sifts corresponding to gridID = x we record topOfBucket 
	// +---------------+---+---+---+---+---+---+---+---
	// |GridID         | 0 | 1 | 2 | 3 | 4 | 5 | 6 | ...
	// +---------------+---+---+---+---+---+---+---+---
	// |TopOfBucket    | 0 |-1 | 2 |-1 |-1 | 3 | 6 | ...
	// +---------------+---+---+---+---+---+---+---+---

	// filhal
	hipMemcpy(d_gridID, d_gridID1, sizeof(int)*target.num_keys, hipMemcpyDeviceToDevice);
	hipMemcpy(d_target_siftID, d_target_siftID1, sizeof(int)*target.num_keys, hipMemcpyDeviceToDevice);

	//================== Record topOfBucket =====================

	clock_gettime(CLOCK_MONOTONIC, &t1);

	thrust::device_vector<int> d_topOfBucket(target_grid.numGrids, -1);
	thrust::device_vector<int> d_numSift(target_grid.numGrids, 0);

	thrust::device_vector<int> d_topOfBucket2(target_grid.numGridsXOv, -1);
	thrust::device_vector<int> d_numSift2(target_grid.numGridsXOv, 0);

	thrust::device_vector<int> d_topOfBucket3(target_grid.numGridsYOv, -1);
	thrust::device_vector<int> d_numSift3(target_grid.numGridsYOv, 0);

	thrust::device_vector<int> d_topOfBucket4(target_grid.numGridsXYOv, -1);
	thrust::device_vector<int> d_numSift4(target_grid.numGridsXYOv, 0);

	int *d_topOfBucket_ptr = thrust::raw_pointer_cast(d_topOfBucket.data());
	int *d_numSift_ptr = thrust::raw_pointer_cast(d_numSift.data());

	int *d_topOfBucket2_ptr = thrust::raw_pointer_cast(d_topOfBucket2.data());
	int *d_numSift2_ptr = thrust::raw_pointer_cast(d_numSift2.data());

	int *d_topOfBucket3_ptr = thrust::raw_pointer_cast(d_topOfBucket3.data());
	int *d_numSift3_ptr = thrust::raw_pointer_cast(d_numSift3.data());

	int *d_topOfBucket4_ptr = thrust::raw_pointer_cast(d_topOfBucket4.data());
	int *d_numSift4_ptr = thrust::raw_pointer_cast(d_numSift4.data());

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_thrust2 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//==================  Call the GPU kernel  ====================

	clock_gettime(CLOCK_MONOTONIC, &t1);

	findTopOfBucket<<<numBlocks, threadsPerBlock>>>(d_gridID,d_gridID2,d_gridID3,d_gridID4, target.num_keys,d_topOfBucket_ptr,d_topOfBucket2_ptr,d_topOfBucket3_ptr,d_topOfBucket4_ptr,target_grid.numGrids,target_grid.numGridsXOv,target_grid.numGridsYOv,target_grid.numGridsXYOv);
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_topOfBucket = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//==============================================================

	//========== Record the number of sifts in each grid ===========  

	threadsPerBlock = 512;
	numBlocks = ceil((float)(max(target_grid.numGrids,target_grid.numGridsXOv,target_grid.numGridsYOv,target_grid.numGridsXYOv))/threadsPerBlock);

	clock_gettime(CLOCK_MONOTONIC, &t1);

	findNumSift<<<numBlocks, threadsPerBlock>>>(d_gridID,d_topOfBucket_ptr,d_gridID2,d_topOfBucket2_ptr,d_gridID3,d_topOfBucket3_ptr,d_gridID4,d_topOfBucket4_ptr,target_grid.numGrids,target_grid.numGridsXOv,target_grid.numGridsYOv,target_grid.numGridsXYOv,d_numSift_ptr,d_numSift2_ptr,d_numSift3_ptr,d_numSift4_ptr);
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_numSift= ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);


	//==============================================================


	//====================  CLUSTERING STEP ========================
	// This step deals with finding the epipolar lines for each sift 
	// of the source image and clustering sifts with similar epipolar
	// lines together.
	//==============================================================


	//=================== Find epipolar lines ======================
	// The kernel findEpipolarLine finds the epipolar line and stores 
	// stores its points of intersection with the boundaries of the  
	// target image in d_epipolarPoints.
	//==============================================================

	//=============== Decide kernel configuration ==================

	threadsPerBlock = 512;
	numBlocks = ceil(((float)source.num_keys)/threadsPerBlock);

	clock_gettime(CLOCK_MONOTONIC, &t1);

	findEpipolarLine<<<numBlocks, threadsPerBlock>>>(d_fmatrix,d_keys1,source.width,source.height, target.width, target.height,d_epipolarPoints,source.num_keys);	
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_findEpiline = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//==== Sort the epipolar lines (keys) with siftIDs as values ====

	clock_gettime(CLOCK_MONOTONIC, &t1);

	thrust::device_ptr<long long int> d_epipolarPoints_ptr(d_epipolarPoints);
	thrust::device_ptr<int> d_source_siftID_ptr(d_source_siftID);

	thrust::sort_by_key(d_epipolarPoints_ptr, d_epipolarPoints_ptr+source.num_keys,d_source_siftID_ptr);

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_thrust4 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);


	//==============================================================

	// Current scene :
	// sifts with similar epipolar lines are together now
	// +----------------------+---+---+---+---+---+---+---+----
	// |Epipolar Line end pts | a | b | c | d | e | f | g | ... (sorted)
	// +----------------------+---+---+---+---+---+---+---+----
	// |SiftID                | m | n | o | p | q | r | s | ... (random order)
	// +----------------------+---+---+---+---+---+---+---+----
	// An array called cluster(h_cluster and d_cluster) will set the boundaries
	// If epl[i] - epl[i-1] < threshold then cluster[i] = 0 else cluster[i] = 1
	//==============================================================

	h_cluster[0] = 0;

	clock_gettime(CLOCK_MONOTONIC, &t1);

	hipMemcpy(d_cluster, h_cluster, sizeof(int)*source.num_keys, hipMemcpyHostToDevice); 

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_memcpy3 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//=== Cluster the epipolarPoints together (similar epipolar lines in one group) ===

	clock_gettime(CLOCK_MONOTONIC, &t1);

	clusterLines<<<numBlocks, threadsPerBlock>>>(d_epipolarPoints,d_cluster,source.num_keys);
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_clusterLines = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	// Scene after clustering :
	// sifts with similar epipolar lines are together now
	// +----------------------+---+---+---+---+---+---+---+----
	// |Epipolar Line end pts | a | b | c | d | e | f | g | ... (sorted)
	// +----------------------+---+---+---+---+---+---+---+----
	// |SiftID                | m | n | o | p | q | r | s | ... (random order)
	// +----------------------+---+---+---+---+---+---+---+----
	// |cluster               | 0 | 0 | 1 | 0 | 1 | 1 | 1 | ... 
	// +----------------------+---+---+---+---+---+---+---+----
	// Now compute prefix sum to assign cluster ids
	//==============================================================


	clock_gettime(CLOCK_MONOTONIC, &t1);

	thrust::device_ptr<int> d_cluster_ptr(d_cluster);

	thrust::inclusive_scan(d_cluster_ptr, d_cluster_ptr + source.num_keys, d_cluster_ptr);	

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_thrust5 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//================= current scene example ======================
	//
	// +---------------+---+---+---+---+---+---+---+----
	// |id             | 0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 ...
	// +---------------+---+---+---+---+---+---+---+----
	// |siftID         | a | b | c | d | e | f | g | h ...
	// +---------------+---+---+---+---+---+---+---+----
	// |h_cluster      | 0 | 1 | 1 | 2 | 3 | 3 | 3 | 4 ...
	// +---------------+---+---+---+---+---+---+---+----
	//
	//  so we want cluster location to be like this
	//
	// +-----------------+---+---+---+---+---+---+---+----
	// |cluster_id       | 0 | 1 | 2 | 3 | 4 | ...
	// +-----------------+---+---+---+---+---+---+---+----
	// |cluster_location | 0 | 1 | 3 | 4 | 7 | ...
	// +-----------------+---+---+---+---+---+---+---+----
	//
	//==============================================================

	clock_gettime(CLOCK_MONOTONIC, &t1);

	hipMemcpy(h_cluster, d_cluster, sizeof(int)*source.num_keys, hipMemcpyDeviceToHost); 

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_memcpy4 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//==============================================================
	// =================  Record Number of Clusters ================

	int numClusters = h_cluster[source.num_keys-1]+1;

	//==============================================================

	clock_gettime(CLOCK_MONOTONIC, &t1);

	thrust::device_vector<int> d_clusterLocation(numClusters, -1);
	d_clusterLocation[h_cluster[0]] = 0;

	int * d_clusterLocation_ptr = thrust::raw_pointer_cast(d_clusterLocation.data());

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_thrust6 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//==============================================================

	//==================== Find Cluster Location ===================

	clock_gettime(CLOCK_MONOTONIC, &t1);

	findClusterLocation<<<numBlocks, threadsPerBlock>>>(d_cluster, source.num_keys,d_clusterLocation_ptr,d_source_siftID,d_clusterID);
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_findClusterLocation = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//==============================================================

	//===================== Matching Step ==========================
	//
	//    Matching involves the following two kernels : 
	//    -----------------------------------------------------------
	//
	// -> FindPotentialMatches which finds eqidistant points on the 
	//    epipolar line and the grids those points fall into. Then it
	//    stores the target sifts which are a part of those grids.
	//    These sifts are our potential matches. This is done for each 
	//    cluster.
	//
	//    -----------------------------------------------------------
	//
	// -> FindMatches which, for each source sift, loads the potential
	//    matches and computes the euclidian distances. It then takes 
	//    the ratio of minimum and next minimum distance and declares 
	//    a match based on some criterias.
	//	
	//==============================================================

// [date : 1 oct 2015] first we do a find potential matches to just find the num of potential matches for each cluster
// then we find the max of these num_potential_matches
// and then we malloc the d_pot_patches array with that size 

	//================= Decide kernel configuration ================

	threadsPerBlock = 256 ;
	numBlocks = numClusters;

	//==============================================================
	thrust::device_vector<int> d_numPotentialMatches(numClusters, 0);

	int * d_numPotentialMatches_ptr = thrust::raw_pointer_cast(d_numPotentialMatches.data());


	clock_gettime(CLOCK_MONOTONIC, &t1);

	findNumPotentialMatchesForEachCluster<<<numBlocks, threadsPerBlock, 2*threadsPerBlock*sizeof(int)>>>(d_gridID,d_target_siftID,d_topOfBucket_ptr,d_gridID2,d_target_siftID2,d_topOfBucket2_ptr,d_gridID3,d_target_siftID3,d_topOfBucket3_ptr,d_gridID4,d_target_siftID4,d_topOfBucket4_ptr,target_grid.gridSize,d_clusterLocation_ptr,d_epipolarPoints,d_numSift_ptr,d_numSift2_ptr,d_numSift3_ptr,d_numSift4_ptr,target.height, target_grid.numXGrids1,target_grid.halfSize,target_grid.numXGrids2,target_grid.numGridsXOv,target_grid.numGridsYOv,target_grid.numGrids,target_grid.numGridsXYOv,d_numPotentialMatches_ptr);	
	hipDeviceSynchronize();
	
//	for(int i = 0 ; i < d_numPotentialMatches.size(); i++)
//		std::cout << d_numPotentialMatches[i] << std::endl;

	thrust::device_vector<int>::iterator d_max = thrust::max_element(d_numPotentialMatches.begin(),d_numPotentialMatches.end());
//	int *result = thrust::min_element(d_numPotentialMatches.begin(), d_numPotentialMatches.end());
	int maxNumPotentialMatches = *d_max;

	printf("maxpot = %d\n",maxNumPotentialMatches);


	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_computeNumPotentialMatches = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);



	//================ Malloc d_potentialMatches ===================

	clock_gettime(CLOCK_MONOTONIC, &t1);

	int *d_potentialMatches;
	hipMalloc((void **)&d_potentialMatches,numClusters*(maxNumPotentialMatches+2)*sizeof(int));	// allocate memory on device

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_malloc4 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//==============================================================



	//=========== Find Potential Matches for each cluster ==========

	clock_gettime(CLOCK_MONOTONIC, &t1);

	findPotentialMatchesForEachCluster<<<numBlocks, threadsPerBlock, 2*threadsPerBlock*sizeof(int)>>>(d_gridID,d_target_siftID,d_topOfBucket_ptr,d_gridID2,d_target_siftID2,d_topOfBucket2_ptr,d_gridID3,d_target_siftID3,d_topOfBucket3_ptr,d_gridID4,d_target_siftID4,d_topOfBucket4_ptr,target_grid.gridSize,d_clusterLocation_ptr,d_epipolarPoints,d_numSift_ptr,d_numSift2_ptr,d_numSift3_ptr,d_numSift4_ptr,d_potentialMatches,target.height, target_grid.numXGrids1,target_grid.halfSize,target_grid.numXGrids2,target_grid.numGridsXOv,target_grid.numGridsYOv,target_grid.numGrids,target_grid.numGridsXYOv,maxNumPotentialMatches);	
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_computePotentialMatches = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//==============================================================

	// Declare matches array which stores the matching siftID 
	// ( or -1 if no match found) for each query sift 

	clock_gettime(CLOCK_MONOTONIC, &t1);

	int *match = (int *)malloc(sizeof(int)*source.num_keys);
	int *d_matches_ptr;
	hipMalloc((void**)&d_matches_ptr, sizeof(int)*source.num_keys);
	for(i = 0 ; i < source.num_keys; i++)
	{
		match[i] = -1;
	}
	hipMemcpy(d_matches_ptr, match, sizeof(int)*source.num_keys, hipMemcpyHostToDevice); 

	int *h_potentialMatches = (int *)malloc(numClusters*(2+maxNumPotentialMatches)*sizeof(int));	// allocate memory on device
	hipMemcpy(h_potentialMatches , d_potentialMatches,numClusters*(maxNumPotentialMatches+2)*sizeof(int) , hipMemcpyDeviceToHost); 
	/*
	for(int i = 0 ; i < numClusters; i++)
	{
		if(h_potentialMatches[maxNumPotentialMatches*i] != 0)
			printf("numpotm = %d\n",h_potentialMatches[maxNumPotentialMatches*i]);
	}
*/

//	thrust::device_vector<int> d_matches(source.num_keys, -1);
//	int * d_matches_ptr = thrust::raw_pointer_cast(d_matches.data());

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_thrust3 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//==============================================================

	//================= Decide kernel configuration ================

	threadsPerBlock = 256 ;
	numBlocks = source.num_keys;

	//==============================================================

	//======================== Find Matches ========================

	clock_gettime(CLOCK_MONOTONIC, &t1);

	findMatches<<<numBlocks, threadsPerBlock,4*threadsPerBlock*sizeof(int)>>>(d_matches_ptr,d_clusterID,d_potentialMatches,maxNumPotentialMatches);
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_computeMatches = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);
	//==============================================================

	hipMemcpy(match, d_matches_ptr, sizeof(int)*source.num_keys, hipMemcpyDeviceToHost); 
	//======================  memcpy  ==============================

	clock_gettime(CLOCK_MONOTONIC, &t1);

//	thrust::host_vector<int> h_matches = d_matches;

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_memcpy5 = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);


	//======================== CLEAN UP ============================

	//===================== bind the textures ====================

	clock_gettime(CLOCK_MONOTONIC, &t1);

	hipUnbindTexture (tex_targetDescriptor);
	hipUnbindTexture (tex_sourceDescriptor);

	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_unbindtex = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	//==============================================================

	//===================== Free device memory =====================

	clock_gettime(CLOCK_MONOTONIC, &t1);

//	hipFree(d_keys1);
//	hipFree(d_keys2);

//	hipFree(d_source_x);
//	hipFree(d_source_y);

//	hipFree(d_target_x);
//	hipFree(d_target_y);

////	hipFree(d_source_keypoints);
////	hipFree(d_target_keypoints);

	hipFree(d_source_siftID);
	hipFree(d_target_siftID);
	hipFree(d_target_siftID1);
	hipFree(d_target_siftID2);
	hipFree(d_target_siftID3);
	hipFree(d_target_siftID4);

	hipFree(d_fmatrix);

	hipFree(d_gridID);
	hipFree(d_gridID1);
	hipFree(d_gridID2);
	hipFree(d_gridID3);
	hipFree(d_gridID4);

	hipFree(d_epipolarPoints);
	hipFree(d_cluster);
	hipFree(d_clusterID);

	hipFree(d_potentialMatches);
	hipFree(d_matches_ptr);
	//======================== Free host memory ====================

/*	free(h_target_siftID);
	free(h_gridID);
	free(target_x);
	free(target_y);
	free(target_scale);
	free(target_orient);
	free(target_keypoints);
	free(h_source_siftID);
	free(source_x);
	free(source_y);
	free(source_scale);
	free(source_orient);
	free(source_keypoints);
*/	free(h_cluster);
	free(h_epipolarPoints);
	free(h_gridID);
	free(h_potentialMatches);
	clock_gettime(CLOCK_MONOTONIC, &t2);
	double time_freemem = ((t2.tv_sec - t1.tv_sec)*1000) + (((double)(t2.tv_nsec - t1.tv_nsec))/1000000.0);

	clock_gettime(CLOCK_MONOTONIC, &t4);
	double time_total = ((t4.tv_sec - t3.tv_sec)*1000) + (((double)(t4.tv_nsec - t3.tv_nsec))/1000000.0);
	double time_gpu = ((t4.tv_sec - t5.tv_sec)*1000) + (((double)(t4.tv_nsec - t5.tv_nsec))/1000000.0);
	//==============================================================

	int num_matches = 0;

	for(i = 0 ; i < source.num_keys; i++)
	{
		if(match[i]!=-1)
		{
			num_matches++;
			matches.push_back(std::make_pair(i,match[i]));
		}
	}
	free(match);

	//==============================================================
  /*
	std::cout << "Time on GPU init: " << time_gpu_init << std::endl;	
	std::cout << "Time on CPU setup: " << time_total_cpu_alloc << std::endl;	
	std::cout << "Time - memcpy : " << time_memcpy1+time_memcpy3+time_memcpy4+time_memcpy5 << std::endl;
	printf("time_memcpy1 %lf, time_memcpy3 %lf , time_memcpy4 %lf, time_memcpy5 %lf\n",time_memcpy1,time_memcpy3,time_memcpy4,time_memcpy5);
	std::cout << "Time - malloc : " << time_malloc1+time_malloc4 << std::endl;
	std::cout << "Time to compute gridID : " << time_computeGridID << std::endl;
	std::cout << "Time - thrust : " << time_thrust1+time_thrust2+time_thrust3+time_thrust4+time_thrust5+time_thrust6 << std::endl;
	std::cout << "Time to compute topOfBucket : " << time_topOfBucket << std::endl;
	std::cout << "Time to find num sifts per grid : " << time_numSift << std::endl;
	std::cout << "Time to find epipolar lines : " << time_findEpiline << std::endl;
	std::cout << "Time to cluster lines : " << time_clusterLines << std::endl;
	std::cout << "Time to find cluster location : " << time_findClusterLocation << std::endl;
	
	std::cout << "Time to compute num potential matches per cluster: " << time_computeNumPotentialMatches << std::endl;
	std::cout << "Time to compute potential matches per cluster: " << time_computePotentialMatches << std::endl;
	std::cout << "Time to compute matches: " << time_computeMatches << std::endl;
	std::cout << "Time to unbind textures : " << time_unbindtex << std::endl;
	std::cout << "Time to free memory : " << time_freemem << std::endl;
	std::cout << "total time on gpu: " << time_gpu << std::endl;
	std::cout << "total time : " << time_total << std::endl;
*/
	std::cout << "Total matches found : " << num_matches << std::endl;

	std::cout << time_gpu << std::endl;
	return time_gpu ;




	//==============================================================

}
