#include "hip/hip_runtime.h"
#include "defs.h"
#include "Gridder.h"
#include "sifts.h"

void Grid::g_init(sift_img h_query)
{

	int num_keys = h_query.num_keys;
	gridSize = 8;	
	imageWidth = h_query.width;
	imageHeight = h_query.height;

	if(gridSize%2 == 1) gridSize++;
	
	halfSize = gridSize/2;

	numXGrids1 = (int)ceil((float)imageWidth/gridSize);
	numYGrids1 = (int)ceil((float)imageHeight/gridSize);
	numXGrids2 = (int)ceil((float)(imageWidth-halfSize)/gridSize);
	numYGrids2 = (int)ceil((float)(imageHeight-halfSize)/gridSize);

	numGrids = numXGrids1*numYGrids1;
	numGridsXOv = numXGrids2*numYGrids1;
	numGridsYOv = numXGrids1*numYGrids2;
	numGridsXYOv = numXGrids2*numYGrids2;
	//	int numGridsXOv = numXGrids2*numYGrids1;
	//	int numGridsYOv = numXGrids1*numYGrids2;
	//	int numGridsXYOv = numXGrids2*numYGrids2;

	ov = (float)halfSize/(float)gridSize;

//==============================================================


}
